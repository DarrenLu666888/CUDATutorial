#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    // 初始化 CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA 初始化失败: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // 获取 GPU 设备数量
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "未找到支持 CUDA 的 GPU 设备！" << std::endl;
        return 1;
    }

    // 遍历所有 GPU 设备
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        // 输出计算能力
        std::cout << "GPU " << i << ": " << deviceProp.name << std::endl;
        std::cout << "计算能力: sm_" << deviceProp.major<<"(major), sm_" << deviceProp.minor <<"(minor)"<< std::endl;
        std::cout << "maxBlockSize: " << deviceProp.maxBlockSize[0] << std::endl;
        std::cout << "-----------------------------" << std::endl;
    }

    return 0;
}